#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "snn_library.h"
#include "neuron_models/GPU_lif_neuron.cuh"

#define THR_PER_BLOCK 1024 

/**
D = A * B + C kalkulua egiten duen kernela
*/
__global__ void cuda_add_dot_matrix(int rowsAC, int colsBC, int colsArowsB, float *A, float *B, float *C, float *D)
{
    //lortu hariaren identifikadorea
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

    //hariak kalkulatu behar duen Dko elementuaren errenkada eta zutabea
    int zutab = i%colsBC;
    int errenk = i/colsBC;

    //egiaztatu hariak kalkulua egin behar duela
    if(i<(rowsAC * colsBC))
    {
        //D kalkulatzeko Ako errenkada eta Bko zutabea prozesatu
        for(j=0; j<colsArowsB; j++)  
            D[i]+=A[errenk * colsArowsB + j]*B[j * colsBC + zutab];

        D[i] += C[i];
    }
}


__global__ void cuda_simulation_step_lif_neuron(int rowsAC, int colsBC, int colsArowsB, float *A, float *B, float *C, float *D)
{
    //lortu hariaren identifikadorea
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

    //hariak kalkulatu behar duen Dko elementuaren errenkada eta zutabea
    int zutab = i%colsBC;
    int errenk = i/colsBC;

    //egiaztatu hariak kalkulua egin behar duela
    if(i<(rowsAC * colsBC))
    {
        //D kalkulatzeko Ako errenkada eta Bko zutabea prozesatu
        for(j=0; j<colsArowsB; j++)  
            D[i]+=A[errenk * colsArowsB + j]*B[j * colsBC + zutab];

        D[i] += C[i];
    }
}


void simulate_in_GPU(spiking_nn_t *snn, simulation_configuration_t *conf, simulation_results_t *results){

    // copy data to GPU
    copy_snn_structure_to_GPU(snn);

    // simulate
}


double copy_snn_structure_to_GPU(spiking_nn_t *snn){
    
    // event variables
    hipEvent_t start, stop;
    
    // structs for snn, neurons and synapses
    spiking_nn_t *d_snn;
    lif_neuron_t *d_lif_neurons; 
    synapse_t *d_synapses;

    // control variables
    int i, j;

    // cuda things
    float milliseconds = 0;
    int thr_per_blk_neurons, blk_in_grid_neurons, thr_per_blk_synapses, blk_in_grid_synapses;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // reserve memory for neurons and synapses lists
    //gpuErrchk(hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t)));
    //gpuErrchk(hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t)));



    /* Memory allocation and transfer */

    hipMalloc(&d_snn, sizeof(spiking_nn_t)); // allocate memory for neurons
    hipMemcpy(d_snn, snn, sizeof(spiking_nn_t), hipMemcpyHostToDevice); // copy neurons information

    hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t)); // allocate memory for neurons
    hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice); // copy neurons information

    hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t)); // allocate memory for synapses
    hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice); // copy synapses information



    d_snn->lif_neurons = d_lif_neurons;
    d_snn->synapses = d_synapses;
    // TODO: connect neuron initializer... if it is necessary


    // allocate memory for each neuron elements
    for(i = 0; i<snn->n_neurons; i++){
        //gpuErrchk(hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int)));
        
        // allocate memory for input and outpyt synapse indexes, and copy
        hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int));
        hipMalloc(&d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int));
        hipMemcpy(&d_lif_neurons[i].input_synapse, snn->lif_neurons[i].input_synapse, snn->lif_neurons[i].n_input_synapse * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&d_lif_neurons[i].output_synapse, snn->lif_neurons[i].output_synapse, snn->lif_neurons[i].n_output_synapse * sizeof(int), hipMemcpyHostToDevice);

        // allocate memory for next spike indexes and initialize
        hipMalloc(&d_lif_neurons[i].next_synapse_index, snn->lif_neurons[i].n_input_synapse * sizeof(int));     
        for(j = 0; j<lif_neurons[i].n_input_synapse; j++){
            d_lif_neurons[i].next_synapse_index[j] = 0;
        }  

        // allocate memory for spike times and initialize
        hipMalloc(&d_lif_neurons[i].spike_times_arr, snn->lif_neurons[i].max_spikes * sizeof(int));
        for(j = 0; j<lif_neurons[i].max_spikes; j++){
            d_lif_neurons[i].spike_times_arr[j] = -1;
        }  
    }


    /* initialize references (pointers) */
    
    // connect neurons and synapses
    lif_neuron_t *d_neuron;
    synapse_t *d_synapse;
    for(i=0; i<snn->n_neurons; i++){

        d_neuron = &(d_neurons[i]);
        for(j=0; j<snn->lif_neurons[i].n_input_synapse){

            d_synapse = &(d_synapses[snn->lif_neurons[i].input_synapses[j]]);
            d_synapse->post_synaptic_lif_neuron = d_neuron;
        }

        for(j=0; j<snn->lif_neurons[i].n_output_synapse){

            d_synapse = &(d_synapses[snn->lif_neurons[i].output_synapses[j]]);
            d_synapse->pre_synaptic_lif_neuron = d_neuron;
        }
    }
    
    // set training rule for synapse // TODO: this must be refactorized to a function
    synapse->lr = lists->training_zones[synapse_id];

    for(i=0; i<snn.n_synapses; i++){
        synapse_t *d_synapse = &(d_synapses[i]);

        switch (d_synapse->lr) // get synapse training zone from list
        {
            case 0:
                d_synapse->learning_rule = &add_stdp;//(void (*)())&add_stdp;
                break;
            case 1:
                d_synapse->learning_rule = &mult_stdp;//(void (*)())&mult_stdp;
                break;
            case 2:
                d_synapse->learning_rule = &anti_stdp;//(void (*)())&anti_stdp;
                break;
            //case 3:
            //    synapse->learning_rule = &triplet_stdp;//(void (*)())&triplet_stdp;
            //    break;*/
            default:
                d_synapse->learning_rule = &add_stdp;//(void (*)())&add_stdp;
                break;
        }
    }
    


    /* Copy information from CPU to GPU */

    // copy information to gpu
    //gpuErrchk(hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice));

    // copy neurons information

    // copy synapses information

    // copy info of neurons (ONLY POINTERS; HOW IS THE REST OF INFORMATION PASSED?)
    for(int i = 0; i<snn->n_neurons; i++){
        hipMemcpy(d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int), hipMemcpyHostToDevice);
   }


    return 0.0;    
}


/**
GPUko memoriara mugitu matrizeak eta jaurti kernela
*/
double process_simulation_lif_neuron(spiking_nn_t *snn, int n, int m, int time_steps){
    printf("Running CUDA code, YUJUUUU\n");
    
    hipEvent_t start, stop;
    
    // list of neurons and synapses
    lif_neuron_t *d_lif_neurons; 
    synapse_t *d_synapses;

    // cuda things
    float milliseconds = 0;
    int thr_per_blk_neurons, blk_in_grid_neurons, thr_per_blk_synapses, blk_in_grid_synapses;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // reserve memory for neurons and synapses lists
    //gpuErrchk(hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t)));
    //gpuErrchk(hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t)));
    hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t));
    hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t));

    // reserve memory for each neuron synapse list indexes
    for(int i = 0; i<snn->n_neurons; i++){
        //gpuErrchk(hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int)));
        hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int));
        //gpuErrchk(hipMalloc(&d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int)));
        hipMalloc(&d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int));
    }

    // reserve memory for synapse pointers
    for(int i = 0; i<snn->n_synapses; i++){
        //gpuErrchk(hipMalloc(&d_synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int)));
        hipMalloc(&d_synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int));
        //gpuErrchk(hipMalloc(&d_synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t)));
        hipMalloc(&d_synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t));
        //gpuErrchk(hipMalloc(&d_synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t)));
        hipMalloc(&d_synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t));
        //d_synapse[i].learning_rule = snn->synapses[i].learning_rule;
    }


    // copy information to gpu
    //gpuErrchk(hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice));
    hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice);
    hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice);

    // copy info of neurons (ONLY POINTERS; HOW IS THE REST OF INFORMATION PASSED?)
    for(int i = 0; i<snn->n_neurons; i++){
        hipMemcpy(d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int), hipMemcpyHostToDevice);
   }

    // reserve memory for synapse pointers
    for(int i = 0; i<snn->n_synapses; i++){
        hipMemcpy(&d_synapses[i].l_spike_times, snn->synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&d_synapses[i].pre_synaptic_lif_neuron, snn->synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t), hipMemcpyHostToDevice);
        hipMemcpy(&d_synapses[i].post_synaptic_lif_neuron, snn->synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t), hipMemcpyHostToDevice);
    }

    return 0.0;

    // grid for neurons kernel launching
    //thr_per_blk_neurons = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    //blk_in_grid_neurons = rowsAC; //bloke bakoitzean A-ren errenkada bat

    // grid for synapses kernel launching
    //thr_per_blk_synapses = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    //blk_in_grid_synapses = rowsAC;

    //blokeko gehienez 1024 hari
    /*if(colsBC > 1024)
    {
        int total = rowsAC * colsBC;
        blk_in_grid = total / 1024;
        thr_per_blk = 1024;
    }


    // simulation loop
    //for()
        // launch neuron input synapse kernel
        // launch neuron output synapse kernel
        // launch synapse learning

        // store information?
        
    // launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_add_dot_matrix<<<blk_in_grid, thr_per_blk>>>(rowsAC, colsBC, colsArowsB, d_A, d_B, d_C, d_D);
    gpuErrchk(hipEventRecord(stop));

   //Kopiatu D GPUko memoriatik CPUra
    hipMemcpy(D, d_D, rowsAC * colsBC * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //Askatu GPUko memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return(milliseconds);*/
}


/*double add_dot_matrix_GPU(int rowsAC, int colsBC, int colsArowsB, float* A, float* B, float* C, float* D)
{   
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C, *d_D;
    float milliseconds = 0;
    int thr_per_blk, blk_in_grid;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //erreserbatu memorian GPUan
    gpuErrchk(hipMalloc(&d_A, rowsAC * colsArowsB * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, colsArowsB * colsBC * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, rowsAC * colsBC * sizeof(float)));
    gpuErrchk(hipMalloc(&d_D, rowsAC * colsBC * sizeof(float)));

    //kopiatu A, B eta C matrizeak GPUko memorian
    gpuErrchk(hipMemcpy(d_A, A, rowsAC * colsArowsB * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, colsArowsB * colsBC * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, rowsAC * colsBC * sizeof(float), hipMemcpyHostToDevice));

    //Sareta eta blokeen egitura zehaztu
    thr_per_blk = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    blk_in_grid = rowsAC; //bloke bakoitzean A-ren errenkada bat

    //blokeko gehienez 1024 hari
    if(colsBC > 1024)
    {
        int total = rowsAC * colsBC;
        blk_in_grid = total / 1024;
        thr_per_blk = 1024;
    }

    //jaurti kernela
    gpuErrchk(hipEventRecord(start));
    cuda_add_dot_matrix<<<blk_in_grid, thr_per_blk>>>(rowsAC, colsBC, colsArowsB, d_A, d_B, d_C, d_D);
    gpuErrchk(hipEventRecord(stop));

   //Kopiatu D GPUko memoriatik CPUra
    hipMemcpy(D, d_D, rowsAC * colsBC * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //Askatu GPUko memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return(milliseconds);
}*/

void getProperties(){
    int nDevices;
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
    printf("  minor-major: %d-%d\n", prop.minor, prop.major);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
  }
}
